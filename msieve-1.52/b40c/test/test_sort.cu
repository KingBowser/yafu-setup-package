#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2010 Duane Merrill
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License. 
 * 
 * For more information, see our Google Code project site: 
 * http://code.google.com/p/back40computing/
 * 
 * Thanks!
 ******************************************************************************/


/******************************************************************************
 * Simple test driver program for *large-problem* radix sorting.
 *
 * Useful for demonstrating how to integrate LsbEarlyExit radix sorting into 
 * your application 
 ******************************************************************************/

#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
#include <math.h> 
#include <float.h>
#include <algorithm>

#include <b40c/util/error_utils.cuh>
#include <b40c/util/multi_buffer.cuh>

#include <b40c/radix_sort/enactor.cuh>

#include "b40c_test_util.h"

using namespace b40c;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool 	g_verbose;
bool 	g_keys_only;
int 	g_max_ctas 			= 0;
int 	g_iterations  		= 1;


/******************************************************************************
 * Test structures
 ******************************************************************************/

// Test value-type structure 
struct Fribbitz {
	char a;
	double b;
	unsigned short c;
};


/******************************************************************************
 * Routines
 ******************************************************************************/

/**
 * Displays the commandline usage for this tool
 */
void Usage() 
{
	printf("\ntest_large_problem_sorting [--device=<device index>] [--v] [--i=<num-g_iterations>] "
			"[--max-ctas=<max-thread-blocks>] [--n=<num-elements>] [--keys-only]\n");
	printf("\n");
	printf("\t--v\tDisplays sorted results to the console.\n");
	printf("\n");
	printf("\t--i\tPerforms the sorting operation <num-g_iterations> times\n");
	printf("\t\t\ton the device. Re-copies original input each time. Default = 1\n");
	printf("\n");
	printf("\t--n\tThe number of elements to comprise the sample problem\n");
	printf("\t\t\tDefault = 512\n");
	printf("\n");
	printf("\t--keys-only\tSpecifies that keys are not accommodated by value pairings\n");
	printf("\n");
}



/**
 * Uses the GPU to sort the specified vector of elements for the given
 * number of g_iterations, displaying runtime information.
 */
template <
	radix_sort::ProblemSize GENRE,
	typename PingPongStorage,
	typename SizeT>
void TimedSort(
	PingPongStorage 						&device_storage,
	SizeT 									num_elements,
	typename PingPongStorage::KeyType 		*h_keys,
	int 									g_iterations)
{
	typename PingPongStorage::KeyType K;

	// Create sorting enactor
	radix_sort::Enactor sorting_enactor;

	// Move a fresh copy of the problem into device storage
	if (util::B40CPerror(hipMemcpy(device_storage.d_keys[device_storage.selector], h_keys, sizeof(K) * num_elements, hipMemcpyHostToDevice),
		"TimedSort hipMemcpy device_storage.d_keys[0] failed: ", __FILE__, __LINE__)) exit(1);

	// Perform a single sorting iteration to allocate memory, prime code caches, etc.
	sorting_enactor.Sort<GENRE, sizeof(K) * 8, 0>(device_storage, 
						num_elements, 0, g_max_ctas, true);

	// Perform the timed number of sorting g_iterations
	GpuTimer timer;

	double elapsed = 0;
	for (int i = 0; i < g_iterations; i++) {

		// Move a fresh copy of the problem into device storage
		if (util::B40CPerror(hipMemcpy(device_storage.d_keys[device_storage.selector], h_keys, sizeof(K) * num_elements, hipMemcpyHostToDevice),
			"TimedSort hipMemcpy device_storage.d_keys[0] failed: ", __FILE__, __LINE__)) exit(1);

		// Start cuda timing record
		timer.Start();

		// Call the sorting API routine
		sorting_enactor.Sort<GENRE, sizeof(K) * 8, 0>(device_storage, 
							num_elements, 0, g_max_ctas);

		// End cuda timing record
		timer.Stop();
		elapsed += (double) timer.ElapsedMillis();
	}

	// Display timing information
	double avg_runtime = elapsed / g_iterations;
	double throughput = ((double) num_elements) / avg_runtime / 1000.0 / 1000.0;
    printf(", %f GPU ms, %f x10^9 elts/sec\n", 
		avg_runtime,
		throughput);
	
    // Copy out data
    if (util::B40CPerror(hipMemcpy(h_keys, device_storage.d_keys[device_storage.selector], sizeof(K) * num_elements, hipMemcpyDeviceToHost),
		"TimedSort hipMemcpy device_storage.d_keys failed: ", __FILE__, __LINE__)) exit(1);
}


/**
 * Creates an example sorting problem whose keys is a vector of the specified 
 * number of K elements, values of V elements, and then dispatches the problem 
 * to the GPU for the given number of g_iterations, displaying runtime information.
 *
 * @param[in] 		g_iterations
 * 		Number of times to invoke the GPU sorting primitive
 * @param[in] 		num_elements 
 * 		Size in elements of the vector to sort
 */
template<
	typename K,
	typename V,
	typename SizeT>
void TestSort(SizeT num_elements)
{
    // Allocate the sorting problem on the host and fill the keys with random bytes

	K *h_keys = (K*) malloc(num_elements * sizeof(K));
	K *h_reference_keys = (K*) malloc(num_elements * sizeof(K));
	V *h_values = (g_keys_only) ?
		NULL :
		h_values = (V*) malloc(num_elements * sizeof(V));

	// Use random bits
	for (unsigned int i = 0; i < num_elements; ++i) {
		util::RandomBits<K>(h_keys[i]);
		h_reference_keys[i] = h_keys[i];
	}

	// Run the timing test
	if (g_keys_only) {

		printf("Keys-only, %d iterations, %d elements", g_iterations, num_elements);
		fflush(stdout);

		// Allocate device storage
		util::DoubleBuffer<K> device_storage;
		if (util::B40CPerror(hipMalloc((void**) &device_storage.d_keys[0], sizeof(K) * num_elements),
			"TimedSort hipMalloc device_storage.d_keys[0] failed: ", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipMalloc((void**) &device_storage.d_keys[1], sizeof(K) * num_elements),
			"TimedSort hipMalloc device_storage.d_keys[1] failed: ", __FILE__, __LINE__)) exit(1);

		if (num_elements < 32000) {
			TimedSort<radix_sort::SMALL_PROBLEM>(
				device_storage, num_elements, h_keys, g_iterations);
		}
		else {
			TimedSort<radix_sort::LARGE_PROBLEM>(
				device_storage, num_elements, h_keys, g_iterations);
		}

	    // Free allocated memory
	    if (device_storage.d_keys[0]) hipFree(device_storage.d_keys[0]);
	    if (device_storage.d_keys[1]) hipFree(device_storage.d_keys[1]);

	} else {

		printf("Key-values, %d iterations, %d elements", g_iterations, num_elements);
		fflush(stdout);

		// Allocate device storage
		util::DoubleBuffer<K, V> device_storage;
		if (util::B40CPerror(hipMalloc((void**) &device_storage.d_keys[0], sizeof(K) * num_elements),
			"TimedSort hipMalloc device_storage.d_keys[0] failed: ", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipMalloc((void**) &device_storage.d_values[0], sizeof(V) * num_elements),
			"TimedSort hipMalloc device_storage.d_values[0] failed: ", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipMalloc((void**) &device_storage.d_keys[1], sizeof(K) * num_elements),
			"TimedSort hipMalloc device_storage.d_keys[1] failed: ", __FILE__, __LINE__)) exit(1);
		if (util::B40CPerror(hipMalloc((void**) &device_storage.d_values[1], sizeof(V) * num_elements),
			"TimedSort hipMalloc device_storage.d_values[1] failed: ", __FILE__, __LINE__)) exit(1);

		if (num_elements < 32000) {
			TimedSort<radix_sort::SMALL_PROBLEM>(
				device_storage, num_elements, h_keys, g_iterations);
		}
		else {
			TimedSort<radix_sort::LARGE_PROBLEM>(
				device_storage, num_elements, h_keys, g_iterations);
		}

	    // Free allocated memory
	    if (device_storage.d_keys[0]) hipFree(device_storage.d_keys[0]);
	    if (device_storage.d_keys[1]) hipFree(device_storage.d_keys[1]);
	    if (device_storage.d_values[0]) hipFree(device_storage.d_values[0]);
	    if (device_storage.d_values[1]) hipFree(device_storage.d_values[1]);
	}

	// Flushes any stdio from the GPU
	hipDeviceSynchronize();
    
	// Display sorted key data
	if (g_verbose) {
		printf("\n\nKeys:\n");
		for (int i = 0; i < num_elements; i++) {	
			PrintValue<K>(h_keys[i]);
			printf(", ");
		}
		printf("\n\n");
	}	
	
    // Verify solution
	std::sort(h_reference_keys, h_reference_keys + num_elements);	
	CompareResults<K>(h_keys, h_reference_keys, num_elements, true);
	printf("\n");
	fflush(stdout);

	// Free our allocated host memory 
	if (h_keys != NULL) free(h_keys);
    if (h_values != NULL) free(h_values);
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main(int argc, char** argv)
{
	// Initialize commandline args and device
	CommandLineArgs args(argc, argv);
	b40c::DeviceInit(args);

	// Seed random number generator
	srand(0);				// presently deterministic
	//srand(time(NULL));

	// Use 32-bit integer for array indexing
	typedef int SizeT;
	SizeT num_elements = 1024;

	// Parse command line arguments
    if (args.CheckCmdLineFlag("help")) {
		Usage();
		return 0;
	}

    args.GetCmdLineArgument("i", g_iterations);
    args.GetCmdLineArgument("n", num_elements);
    args.GetCmdLineArgument("max-ctas", g_max_ctas);
    g_keys_only = args.CheckCmdLineFlag("keys-only");
	g_verbose = args.CheckCmdLineFlag("v");

	// Execute test(s)
/*	
	TestSort<float, float>(num_elements);

	TestSort<double, double>(num_elements);

	TestSort<char, char>(num_elements);

	TestSort<unsigned char, unsigned char>(num_elements);

	TestSort<short, short>(num_elements);

	TestSort<unsigned short, unsigned short>(num_elements);

	TestSort<int, int>(

	TestSort<unsigned int, unsigned int>(num_elements);

	TestSort<unsigned long long, unsigned long long>(num_elements);

	TestSort<float, Fribbitz>(num_elements);

	TestSort<unsigned long long, unsigned int>(num_elements);
*/

	TestSort<unsigned long long, unsigned int>(num_elements);
}



