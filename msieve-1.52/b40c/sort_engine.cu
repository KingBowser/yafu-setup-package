#include <stdio.h>
#include <b40c/util/error_utils.cuh>
#include <b40c/util/multi_buffer.cuh>
#include <b40c/radix_sort/enactor.cuh>

#include "sort_engine.h"

typedef unsigned int uint32;

#if defined(_WIN32) || defined (_WIN64)
	#define SORT_ENGINE_DECL __declspec(dllexport)
	typedef unsigned __int64 uint64;
#else
	#define SORT_ENGINE_DECL __attribute__((visibility("default")))
	typedef unsigned long long uint64;
#endif

using namespace b40c;

typedef struct
{
	radix_sort::Enactor enactor;
} sort_engine;

extern "C"
{

SORT_ENGINE_DECL void * 
sort_engine_init(void)
{
	return new sort_engine;
}

SORT_ENGINE_DECL void 
sort_engine_free(void * e)
{
	delete (sort_engine *)e;
}

SORT_ENGINE_DECL void 
sort_engine_run(void * e, sort_data_t * data)
{
	sort_engine *engine = (sort_engine *)e;
	bool need_swap;

	// arrays are assumed packed together; check
	// they would all start on a power-of-two boundary

	if (data->num_arrays > 1 && data->num_elements % 16) {
		printf("sort_engine: invalid array size\n");
		exit(-1);
	}

	if (data->key_bits <= 32) {
		for (size_t i = 0; i < data->num_arrays; i++) {

			hipError_t status;
			util::DoubleBuffer<uint32, uint32> ptrs;

			ptrs.d_keys[0] = (uint32 *)data->keys_in +
						i * data->num_elements;
			ptrs.d_keys[1] = (uint32 *)data->keys_in_scratch +
						i * data->num_elements;
			ptrs.d_values[0] = (uint32 *)data->data_in +
						i * data->num_elements;
			ptrs.d_values[1] = (uint32 *)data->data_in_scratch +
						i * data->num_elements;

			status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		20, 0>(ptrs, data->num_elements,
					data->stream);
			if (status == hipSuccess && data->key_bits > 20) {
				status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		5, 20>(ptrs, data->num_elements,
					data->stream);
			}
			if (status == hipSuccess && data->key_bits > 25) {
				status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		5, 25>(ptrs, data->num_elements,
					data->stream);
			}
			if (status == hipSuccess && data->key_bits > 30) {
				status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		2, 30>(ptrs, data->num_elements,
					data->stream);
			}

			need_swap = (ptrs.selector > 0);
			if (status != hipSuccess) {
				util::B40CPerror(status, "sort engine: ", 
						__FILE__, __LINE__);
				exit(-1);
			}
		}
	}
	else {
		for (size_t i = 0; i < data->num_arrays; i++) {

			hipError_t status;
			util::DoubleBuffer<uint64, uint32> ptrs;

			ptrs.d_keys[0] = (uint64 *)data->keys_in +
						i * data->num_elements;
			ptrs.d_keys[1] = (uint64 *)data->keys_in_scratch +
						i * data->num_elements;
			ptrs.d_values[0] = (uint32 *)data->data_in +
						i * data->num_elements;
			ptrs.d_values[1] = (uint32 *)data->data_in_scratch +
						i * data->num_elements;

			status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		35, 0>(ptrs, data->num_elements,
					data->stream);
			if (status == hipSuccess && data->key_bits > 35) {
				status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		5, 35>(ptrs, data->num_elements,
					data->stream);
			}
			if (status == hipSuccess && data->key_bits > 40) {
				status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		5, 40>(ptrs, data->num_elements,
					data->stream);
			}
			if (status == hipSuccess && data->key_bits > 45) {
				status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		5, 45>(ptrs, data->num_elements,
					data->stream);
			}
			if (status == hipSuccess && data->key_bits > 50) {
				status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		5, 50>(ptrs, data->num_elements,
					data->stream);
			}
			if (status == hipSuccess && data->key_bits > 55) {
				status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		5, 55>(ptrs, data->num_elements,
					data->stream);
			}
			if (status == hipSuccess && data->key_bits > 60) {
				status = engine->enactor.Sort<
					radix_sort::LARGE_PROBLEM, 
			       		4, 60>(ptrs, data->num_elements,
					data->stream);
			}

			need_swap = (ptrs.selector > 0);
			if (status != hipSuccess) {
				util::B40CPerror(status, "sort engine: ", 
						__FILE__, __LINE__);
				exit(-1);
			}
		}
	}

	if (need_swap == true) {
		std::swap(data->keys_in, data->keys_in_scratch);
		std::swap(data->data_in, data->data_in_scratch);
	}
}

} // extern "C"
